#include "cuASR_helper.h"

#include "cuasr/gemm/device/default_srgemm_configuration.h"
#include "cuasr/gemm/device/srgemm.h"
#include "cuasr/functional.h"

#include <cstdlib>
#include <experimental/source_location>
#include <iostream>

namespace {
void check_for_cuda_error(
    [[maybe_unused]] std::experimental::source_location s = std::experimental::source_location::current()) {
#ifndef NDEBUG
    hipError_t err = hipGetLastError();
    if ( err != hipSuccess ) {
        std::cerr << "CUDA Error: " <<  hipGetErrorString(err) << '\n';
        std::cerr << "    file: " << s.file_name() << '\n'
                  << "    function: " << s.function_name() << '\n'
                  << "    line: " << s.line() << "\n\n";
        std::exit(1);
    }
#endif
}
}


namespace cuASR_helper {

using AdditionOp       = cuasr::minimum<float>;
using MultiplicationOp = cuasr::plus<float>;

using RowMajor = cutlass::layout::RowMajor;

using cuASR_MinPlus_SGEMM = cuasr::gemm::device::Srgemm<
      AdditionOp, MultiplicationOp,
      HMM::Mod_prob_t, RowMajor,
      HMM::Mod_prob_t, RowMajor,
      HMM::Mod_prob_t, RowMajor,
      HMM::Mod_prob_t
      >;

Dev_mat::Dev_mat(HMM::Mod_prob_t* data, int rows, int cols, size_t bytes_size)
    : data(data), rows(rows), cols(cols), bytes_size(bytes_size)
{}

Dev_mat::Dev_mat(const Dev_mat& rhs) {
    rows = rhs.rows;
    cols = rhs.cols;
    bytes_size = rhs.bytes_size;
    hipMalloc((void **)&data, bytes_size);
    check_for_cuda_error();
    hipMemcpy(rhs.data, data, bytes_size, hipMemcpyDeviceToDevice);
    check_for_cuda_error();
}

Dev_mat::~Dev_mat() {
    hipFree(static_cast<void*>(data));
    check_for_cuda_error();
}

void min_plus_Dev_mat_multiply(const Dev_mat& lhs, const Dev_mat& rhs, Dev_mat& res) {
    if (res.data == nullptr) {
        hipMalloc((void **)&(res.data), sizeof(HMM::Mod_prob_t) * lhs.rows * rhs.cols);
        check_for_cuda_error();
        res.rows = lhs.rows;
        res.cols = rhs.cols;
    }

#ifndef NDEBUG
    if (lhs.cols != rhs.rows) {
        std::cerr << "cuASR: lhs and rhs cols/rows mismatch! "
            << "Lhs.cols is " << lhs.cols << ". "
            << "Rhs.cols is " << rhs.rows << '\n';
    }
    if (lhs.rows != res.rows) {
        std::cerr << "cuASR: lhs and res rows mismatch! "
            << "Lhs.rows is " << lhs.rows << ". "
            << "Res.rows is " << res.rows << '\n';
    }
    if (rhs.cols != res.cols) {
        std::cerr << "cuASR: rhs and res cols mismatch! "
            << "Rhs.cols is " << rhs.cols << ". "
            << "Res.cols is " << res.cols << '\n';
    }
#endif
    auto args = cuASR_MinPlus_SGEMM::Arguments(
        {lhs.rows, lhs.cols, rhs.cols},
        {lhs.data, lhs.rows},
        {rhs.data, rhs.rows},
        {res.data, res.rows},
        {res.data, res.rows},
        {MultiplicationOp::Identity, MultiplicationOp::Annihilator}
    );

    auto minplus_gemm = cuASR_MinPlus_SGEMM();
    auto status = minplus_gemm(args, nullptr, nullptr);
    hipDeviceSynchronize();
    check_for_cuda_error();

    if ((int)status) {
        std::cerr << "Matrix multiply error code " << (int)status << '\n'
            << cutlassGetStatusString(status) << '\n';
    }
}

HMM::Mod_prob_vec_t Dev_mat_to_Prob_vec(const Dev_mat& mat) {
#ifndef NDEBUG
    if (mat.cols != 1) {
        std::cerr << "Error! cuASR Dev_mat is not a column!\n";
        return {};
    }
#endif
    auto host_data = new HMM::Mod_prob_t[mat.rows];
    hipMemcpy(host_data, mat.data, mat.bytes_size, hipMemcpyDeviceToHost);
    check_for_cuda_error();
    auto res = HMM::Mod_prob_vec_t(host_data, host_data + mat.rows);
    delete(host_data);
    return res;
}

void init_matrices_from_HMM(const HMM& hmm, Dev_mat& start_pr, Dev_mat& transp_tr,
    std::vector<Dev_mat>& emit_mat_vec)
{
    // Column for start probs
    auto start_host_ptr = new HMM::Mod_prob_t[hmm.states_num];
    for (size_t i = 0; i < hmm.non_zero_start_probs; ++i) {
        start_host_ptr[hmm.start_probabilities_cols[i]] = hmm.start_probabilities[i];
    }

    // Row major transposed transition matrix
    auto transp_tr_host_ptr = new HMM::Mod_prob_t[hmm.states_num * hmm.states_num];
    for (size_t i = 0; i < hmm.trans_num; ++i) {
        auto row = hmm.trans_cols[i];
        auto col = hmm.trans_rows[i];
        auto val = hmm.trans_probs[i];
        transp_tr_host_ptr[row * hmm.states_num + col] = val;
    }

    // Diagonal matrices
    auto emit_mat_vec_host = std::vector<HMM::Mod_prob_t*>(hmm.emit_num);
    for (size_t i = 0; i < hmm.emit_num; ++i) {
        auto& m = emit_mat_vec_host[i];
        m = new HMM::Mod_prob_t[hmm.states_num * hmm.states_num];
        for (size_t j = 0; j < hmm.states_num; ++j) {
            m[j * hmm.states_num + j] = hmm.emissions[i][j];
        }
    }

    start_pr = Dev_mat {nullptr, (int)hmm.states_num, 1, sizeof(HMM::Mod_prob_t) * hmm.states_num * 1};

    transp_tr = Dev_mat {
        nullptr, (int)hmm.states_num, (int)hmm.states_num, 
        sizeof(HMM::Mod_prob_t) * hmm.states_num * hmm.states_num
    };

    emit_mat_vec = std::vector<Dev_mat>(hmm.emit_num);
    for (size_t i = 0; i < hmm.emit_num; ++i) {
        emit_mat_vec[i] = Dev_mat {
            nullptr, (int)hmm.states_num, (int)hmm.states_num, 
            sizeof(HMM::Mod_prob_t) * hmm.states_num * hmm.states_num
        };
    }


    // Allocate device memory

    hipMalloc((void **)&start_pr.data, start_pr.bytes_size);
    check_for_cuda_error();
    hipMalloc((void **)&transp_tr.data, transp_tr.bytes_size);
    check_for_cuda_error();
    for (size_t i = 0; i < hmm.emit_num; ++i) {
        hipMalloc((void **)&(emit_mat_vec[i].data), emit_mat_vec[i].bytes_size);
        check_for_cuda_error();
    }

    // Transfer data to device
    hipMemcpy(start_pr.data, start_host_ptr, start_pr.bytes_size, hipMemcpyHostToDevice);
    check_for_cuda_error();
    hipMemcpy(transp_tr.data, transp_tr_host_ptr, transp_tr.bytes_size, hipMemcpyHostToDevice);
    check_for_cuda_error();
    for (size_t i = 0; i < hmm.emit_num; ++i) {
        hipMemcpy(emit_mat_vec[i].data, emit_mat_vec_host[i], emit_mat_vec[i].bytes_size, hipMemcpyHostToDevice);
        check_for_cuda_error();
    }

    // Free host memory
    delete(start_host_ptr);
    delete(transp_tr_host_ptr);
    for (auto& m : emit_mat_vec_host) {
        delete(m);
    }
}
} // namespace cuASR_helper
