#include "hip/hip_runtime.h"
#include "cuASR_helper.h"

#include "cuasr/gemm/device/default_srgemm_configuration.h"
#include "cuasr/gemm/device/srgemm.h"
#include "cuasr/functional.h"

#include <algorithm>
#include <cstdlib>
#include <cstring>
#include <experimental/source_location>
#include <iostream>

namespace {
void check_for_cuda_error(
    [[maybe_unused]] std::experimental::source_location s = std::experimental::source_location::current()) {
#ifndef NDEBUG
    hipError_t err = hipGetLastError();
    if ( err != hipSuccess ) {
        std::cerr << "CUDA Error: " <<  hipGetErrorString(err) << '\n';
        std::cerr << "    file: " << s.file_name() << '\n'
                  << "    function: " << s.function_name() << '\n'
                  << "    line: " << s.line() << "\n\n";
        std::exit(1);
    }
#endif
}

void set_to_zero_prob(HMM::Mod_prob_t* data, size_t how_much) {
    for (size_t i = 0; i < how_much; ++i) {
        data[i] = HMM::zero_prob;
    }
}

void copy_Dev_mat(cuASR_helper::Dev_mat& lhs, const cuASR_helper::Dev_mat& rhs) {
    lhs.rows = rhs.rows;
    lhs.cols = rhs.cols;
    lhs.bytes_size = rhs.bytes_size;
    hipMalloc((void **)&(lhs.data), (lhs.bytes_size));
    check_for_cuda_error();
    hipMemcpy(lhs.data, rhs.data, lhs.bytes_size, hipMemcpyDeviceToDevice);
    check_for_cuda_error();
}

void cuda_matrix_deleter(cuASR_helper::Dev_mat& mat) {
    hipFree(static_cast<void*>(mat.data));
    check_for_cuda_error();
}
}


namespace cuASR_helper {

using AdditionOp       = cuasr::minimum<float>;
using MultiplicationOp = cuasr::plus<float>;

using RowMajor = cutlass::layout::RowMajor;

using cuASR_MinPlus_SGEMM = cuasr::gemm::device::Srgemm<
    AdditionOp, MultiplicationOp,
    HMM::Mod_prob_t, RowMajor,
    HMM::Mod_prob_t, RowMajor,
    HMM::Mod_prob_t, RowMajor,
    HMM::Mod_prob_t
    >;

Dev_mat::Dev_mat(HMM::Mod_prob_t* host_data, int rows, int cols, size_t bytes_size)
    : rows(rows), cols(cols), bytes_size(bytes_size)
{
    if (host_data == nullptr) {
        auto host_init_data = new HMM::Mod_prob_t[rows * cols];
        set_to_zero_prob(host_init_data, rows * cols);
        hipMalloc((void **)&data, bytes_size);
        check_for_cuda_error();
        hipMemcpy(data, host_init_data, bytes_size, hipMemcpyHostToDevice);
        check_for_cuda_error();
    } else {
        hipMemcpy(data, host_data, bytes_size, hipMemcpyHostToDevice);
        check_for_cuda_error();
    }
}

Dev_mat::Dev_mat(const Dev_mat& rhs) {
    copy_Dev_mat(*this, rhs);
}

Dev_mat& Dev_mat::operator=(const Dev_mat& rhs) {
    cuda_matrix_deleter(*this);
    copy_Dev_mat(*this, rhs);
    return *this;
}

Dev_mat::~Dev_mat() {
    cuda_matrix_deleter(*this);
}

void min_plus_Dev_mat_multiply(const Dev_mat& lhs, const Dev_mat& rhs, Dev_mat& res) {
    if (res.data == nullptr) {
        res = Dev_mat(nullptr, lhs.rows, rhs.cols, sizeof(HMM::Mod_prob_t) * lhs.rows * rhs.cols);
    }
    #ifndef NDEBUG
    if (lhs.cols != rhs.rows) {
        std::cerr << "cuASR: lhs and rhs cols/rows mismatch! "
            << "Lhs.cols is " << lhs.cols << ". "
            << "Rhs.cols is " << rhs.rows << '\n';
    }
    if (lhs.rows != res.rows) {
        std::cerr << "cuASR: lhs and res rows mismatch! "
            << "Lhs.rows is " << lhs.rows << ". "
            << "Res.rows is " << res.rows << '\n';
    }
    if (rhs.cols != res.cols) {
        std::cerr << "cuASR: rhs and res cols mismatch! "
            << "Rhs.cols is " << rhs.cols << ". "
            << "Res.cols is " << res.cols << '\n';
    }
#endif
    auto args = cuASR_MinPlus_SGEMM::Arguments(
        {res.rows, res.cols, lhs.cols},
        {lhs.data, lhs.cols},
        {rhs.data, rhs.cols},
        {res.data, res.cols},
        {res.data, res.cols},
        {MultiplicationOp::Identity, MultiplicationOp::Annihilator}
    );

    auto minplus_gemm = cuASR_MinPlus_SGEMM();
    auto status = minplus_gemm(args, nullptr, nullptr);
    hipDeviceSynchronize();
    check_for_cuda_error();

    if ((int)status) {
        std::cerr << "Matrix multiply error code " << (int)status << '\n'
            << cutlassGetStatusString(status) << '\n';
    }
}

HMM::Mod_prob_vec_t Dev_mat_to_Prob_vec(const Dev_mat& mat) {
#ifndef NDEBUG
    if (mat.cols != 1) {
        std::cerr << "Error! cuASR Dev_mat is not a column!\n";
    }
#endif
    auto host_data = new HMM::Mod_prob_t[mat.rows * mat.cols];
    hipMemcpy(host_data, mat.data, mat.bytes_size, hipMemcpyDeviceToHost);
    check_for_cuda_error();
    auto res = HMM::Mod_prob_vec_t(host_data, host_data + mat.rows * mat.cols);
    std::replace_if(res.begin(), res.end(), 
        [](auto prob) {
            return HMM::almost_equal(std::numeric_limits<HMM::Mod_prob_t>::max(), prob);
        },
        HMM::zero_prob);
    delete(host_data);
    return res;
}

void init_matrices_from_HMM(const HMM& hmm, Dev_mat& start_pr, Dev_mat& transp_tr,
    std::vector<Dev_mat>& emit_mat_vec)
{
    // Column for start probs
    auto start_host_ptr = new HMM::Mod_prob_t[hmm.states_num];
    set_to_zero_prob(start_host_ptr, hmm.states_num);
    for (size_t i = 0; i < hmm.non_zero_start_probs; ++i) {
        start_host_ptr[hmm.start_probabilities_cols[i]] = hmm.start_probabilities[i];
    }

    // Row major transposed transition matrix
    auto transp_tr_host_ptr = new HMM::Mod_prob_t[hmm.states_num * hmm.states_num];
    set_to_zero_prob(transp_tr_host_ptr, hmm.states_num * hmm.states_num);
    for (size_t i = 0; i < hmm.trans_num; ++i) {
        auto row = hmm.trans_cols[i];
        auto col = hmm.trans_rows[i];
        auto val = hmm.trans_probs[i];
        transp_tr_host_ptr[row * hmm.states_num + col] = val;
    }

    // Diagonal matrices
    auto emit_mat_vec_host = std::vector<HMM::Mod_prob_t*>(hmm.emit_num);
    for (size_t i = 0; i < hmm.emit_num; ++i) {
        auto& m = emit_mat_vec_host[i];
        m = new HMM::Mod_prob_t[hmm.states_num * hmm.states_num];
        set_to_zero_prob(m, hmm.states_num * hmm.states_num);
        for (size_t j = 0; j < hmm.states_num; ++j) {
            m[j * hmm.states_num + j] = hmm.emissions[i][j];
        }
    }

    start_pr = Dev_mat {nullptr, (int)hmm.states_num, 1, sizeof(HMM::Mod_prob_t) * hmm.states_num * 1};

    transp_tr = Dev_mat {
        nullptr, (int)hmm.states_num, (int)hmm.states_num, 
        sizeof(HMM::Mod_prob_t) * hmm.states_num * hmm.states_num
    };

    emit_mat_vec = std::vector<Dev_mat>(hmm.emit_num);
    for (size_t i = 0; i < hmm.emit_num; ++i) {
        emit_mat_vec[i] = Dev_mat {
            nullptr, (int)hmm.states_num, (int)hmm.states_num, 
            sizeof(HMM::Mod_prob_t) * hmm.states_num * hmm.states_num
        };
    }

    // Allocate device memory
    hipMalloc((void **)&start_pr.data, start_pr.bytes_size);
    check_for_cuda_error();
    hipMalloc((void **)&transp_tr.data, transp_tr.bytes_size);
    check_for_cuda_error();
    for (size_t i = 0; i < hmm.emit_num; ++i) {
        hipMalloc((void **)&(emit_mat_vec[i].data), emit_mat_vec[i].bytes_size);
        check_for_cuda_error();
    }

    // Transfer data to device
    hipMemcpy(start_pr.data, start_host_ptr, start_pr.bytes_size, hipMemcpyHostToDevice);
    check_for_cuda_error();
    hipMemcpy(transp_tr.data, transp_tr_host_ptr, transp_tr.bytes_size, hipMemcpyHostToDevice);
    check_for_cuda_error();
    for (size_t i = 0; i < hmm.emit_num; ++i) {
        hipMemcpy(emit_mat_vec[i].data, emit_mat_vec_host[i], emit_mat_vec[i].bytes_size, hipMemcpyHostToDevice);
        check_for_cuda_error();
    }

    // Free host memory
    delete(start_host_ptr);
    delete(transp_tr_host_ptr);
    for (auto& m : emit_mat_vec_host) {
        delete(m);
    }
}
} // namespace cuASR_helper
