#include "hip/hip_runtime.h"
#include "cuASR_helper.h"

#include "cuasr/functional.h"
#include "cuasr/gemm/device/default_srgemm_configuration.h"
#include "cuasr/gemm/device/srgemm.h"

#include <algorithm>
#include <cstdlib>
#include <cstring>
#include <experimental/source_location>
#include <iostream>

namespace {
void check_for_cuda_error([[maybe_unused]] std::experimental::source_location s =
                              std::experimental::source_location::current()) {
#ifndef NDEBUG
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << '\n';
        std::cerr << "    file: " << s.file_name() << '\n'
                  << "    function: " << s.function_name() << '\n'
                  << "    line: " << s.line() << "\n\n";
        std::exit(1);
    }
#endif
}

void set_to_zero_prob(HMM::Mod_prob_t* data, size_t how_much) {
    for (size_t i = 0; i < how_much; ++i) {
        data[i] = HMM::zero_prob;
    }
}

void cuda_matrix_deleter(cuASR_helper::Dev_mat& mat) {
    if (mat.data != nullptr) {
        hipFree(static_cast<void*>(mat.data));
        mat.allocs--;
        check_for_cuda_error();
        mat.data = nullptr;
    }
}

void copy_Dev_mat(cuASR_helper::Dev_mat& lhs, const cuASR_helper::Dev_mat& rhs) {
    cuda_matrix_deleter(lhs);
    lhs.rows = rhs.rows;
    lhs.cols = rhs.cols;
    lhs.bytes_size = rhs.bytes_size;
    hipMalloc((void**)&(lhs.data), lhs.bytes_size);
    check_for_cuda_error();
    lhs.allocs++;
    hipMemcpy((void*)lhs.data, (const void*)rhs.data, lhs.bytes_size, hipMemcpyDeviceToDevice);
    check_for_cuda_error();
}

void move_Dev_mat(cuASR_helper::Dev_mat& lhs, cuASR_helper::Dev_mat&& rhs) {
    cuda_matrix_deleter(lhs);
    lhs.rows = rhs.rows;
    lhs.cols = rhs.cols;
    lhs.bytes_size = rhs.bytes_size;
    lhs.data = rhs.data;
    rhs.data = nullptr;
}
} // namespace

namespace cuASR_helper {

int Dev_mat::allocs = 0;

using AdditionOp = cuasr::minimum<float>;
using MultiplicationOp = cuasr::plus<float>;

using RowMajor = cutlass::layout::RowMajor;

using cuASR_MinPlus_SGEMM =
    cuasr::gemm::device::Srgemm<AdditionOp, MultiplicationOp, HMM::Mod_prob_t, RowMajor,
                                HMM::Mod_prob_t, RowMajor, HMM::Mod_prob_t, RowMajor,
                                HMM::Mod_prob_t>;

Dev_mat::Dev_mat(int rows, int cols)
    : rows(rows), cols(cols), bytes_size(rows * cols * sizeof(HMM::Mod_prob_t)) {
    hipMalloc((void**)&data, bytes_size);
    check_for_cuda_error();
    allocs++;
}

Dev_mat::Dev_mat(const Dev_mat& rhs) : data(nullptr) { copy_Dev_mat(*this, rhs); }

Dev_mat& Dev_mat::operator=(const Dev_mat& rhs) {
    copy_Dev_mat(*this, rhs);
    return *this;
}

Dev_mat::Dev_mat(Dev_mat&& rhs) : data(nullptr) { move_Dev_mat(*this, std::move(rhs)); }

Dev_mat& Dev_mat::operator=(Dev_mat&& rhs) {
    move_Dev_mat(*this, std::move(rhs));
    return *this;
}

Dev_mat::~Dev_mat() { cuda_matrix_deleter(*this); }

void validate_Dev_mat_ptr([[maybe_unused]] const Dev_mat& mat,
                          [[maybe_unused]] const std::string& msg) {
#ifndef NDEBUG
    auto attr = hipPointerAttribute_t();
    hipPointerGetAttributes(&attr, (const void*)mat.data);
    if (attr.memoryType != hipMemoryTypeDevice) {
        std::cout << "Not a device pointer " << msg << ", is host/unregistered? "
                  << (attr.memoryType == hipMemoryTypeHost) << ' '
                  << (attr.memoryType == cudaMemoryTypeUnregistered) << '\n';
    } else {
        std::cout << "OK " << msg << '\n';
    }
#endif
}

void min_plus_Dev_mat_multiply(const Dev_mat& lhs, const Dev_mat& rhs, Dev_mat& res) {
    cuda_matrix_deleter(res);
    res = Dev_mat(lhs.rows, rhs.cols);
    validate_Dev_mat_ptr(res, "res");
#ifndef NDEBUG
    if (lhs.cols != rhs.rows) {
        std::cerr << "cuASR: lhs and rhs cols/rows mismatch! "
                  << "Lhs.cols is " << lhs.cols << ". "
                  << "Rhs.cols is " << rhs.rows << '\n';
    }
    if (lhs.rows != res.rows) {
        std::cerr << "cuASR: lhs and res rows mismatch! "
                  << "Lhs.rows is " << lhs.rows << ". "
                  << "Res.rows is " << res.rows << '\n';
    }
    if (rhs.cols != res.cols) {
        std::cerr << "cuASR: rhs and res cols mismatch! "
                  << "Rhs.cols is " << rhs.cols << ". "
                  << "Res.cols is " << res.cols << '\n';
    }
#endif
    auto args = cuASR_MinPlus_SGEMM::Arguments(
        {res.rows, res.cols, lhs.cols}, {lhs.data, lhs.cols}, {rhs.data, rhs.cols},
        {res.data, res.cols}, {res.data, res.cols},
        {MultiplicationOp::Identity, MultiplicationOp::Annihilator});

    auto minplus_gemm = cuASR_MinPlus_SGEMM();
    auto status = minplus_gemm(args, nullptr, nullptr);
    hipDeviceSynchronize();
    check_for_cuda_error();

    if ((int)status) {
        std::cerr << "Matrix multiply error code " << (int)status << '\n'
                  << cutlassGetStatusString(status) << '\n';
    }
}

HMM::Mod_prob_vec_t Dev_mat_to_Prob_vec(const Dev_mat& mat) {
#ifndef NDEBUG
    if (mat.cols != 1) {
        std::cerr << "Error! cuASR Dev_mat is not a column!\n";
    }
#endif
    auto host_data = new HMM::Mod_prob_t[mat.rows * mat.cols];
    hipMemcpy((void*)host_data, (const void*)mat.data, mat.bytes_size, hipMemcpyDeviceToHost);
    check_for_cuda_error();
    auto res = HMM::Mod_prob_vec_t(host_data, host_data + mat.rows * mat.cols);
    std::replace_if(
        res.begin(), res.end(),
        [](auto prob) {
            return HMM::almost_equal(std::numeric_limits<HMM::Mod_prob_t>::max(), prob);
        },
        HMM::zero_prob);
    delete[] host_data;
    return res;
}

void init_matrices_from_HMM(const HMM& hmm, Dev_mat& start_pr, Dev_mat& transp_tr,
                            std::vector<Dev_mat>& emit_mat_vec) {
    // Column for start probs
    auto start_host_ptr = new HMM::Mod_prob_t[hmm.states_num];
    set_to_zero_prob(start_host_ptr, hmm.states_num);
    for (size_t i = 0; i < hmm.non_zero_start_probs; ++i) {
        start_host_ptr[hmm.start_probabilities_cols[i]] = hmm.start_probabilities[i];
    }

    // Row major transposed transition matrix
    auto transp_tr_host_ptr = new HMM::Mod_prob_t[hmm.states_num * hmm.states_num];
    set_to_zero_prob(transp_tr_host_ptr, hmm.states_num * hmm.states_num);
    for (size_t i = 0; i < hmm.trans_num; ++i) {
        auto row = hmm.trans_cols[i];
        auto col = hmm.trans_rows[i];
        auto val = hmm.trans_probs[i];
        transp_tr_host_ptr[row * hmm.states_num + col] = val;
    }

    // Diagonal matrices
    auto emit_mat_vec_host = std::vector<HMM::Mod_prob_t*>(hmm.emit_num);
    for (size_t i = 0; i < hmm.emit_num; ++i) {
        auto& m = emit_mat_vec_host[i];
        m = new HMM::Mod_prob_t[hmm.states_num * hmm.states_num];
        set_to_zero_prob(m, hmm.states_num * hmm.states_num);
        for (size_t j = 0; j < hmm.states_num; ++j) {
            m[j * hmm.states_num + j] = hmm.emissions[i][j];
        }
    }

    start_pr = Dev_mat((int)hmm.states_num, 1);

    transp_tr = Dev_mat((int)hmm.states_num, (int)hmm.states_num);

    emit_mat_vec = std::vector<Dev_mat>(hmm.emit_num);
    for (size_t i = 0; i < hmm.emit_num; ++i) {
        emit_mat_vec[i] = Dev_mat((int)hmm.states_num, (int)hmm.states_num);
    }

    // Transfer data to device
    hipMemcpy((void*)start_pr.data, (const void*)start_host_ptr, start_pr.bytes_size,
               hipMemcpyHostToDevice);
    check_for_cuda_error();
    hipMemcpy((void*)transp_tr.data, (const void*)transp_tr_host_ptr, transp_tr.bytes_size,
               hipMemcpyHostToDevice);
    check_for_cuda_error();
    for (size_t i = 0; i < hmm.emit_num; ++i) {
        hipMemcpy((void*)emit_mat_vec[i].data, (const void*)emit_mat_vec_host[i],
                   emit_mat_vec[i].bytes_size, hipMemcpyHostToDevice);
        check_for_cuda_error();
    }

    // Free host memory
    delete[] start_host_ptr;
    delete[] transp_tr_host_ptr;
    for (auto& m : emit_mat_vec_host) {
        delete[] m;
    }
}
} // namespace cuASR_helper
